#include "gemm.h"

#include "hipblas.h"

static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}

void gpuBlasSgemm(const float * A, const float *B, float *C, int M, int N, int K, bool no_transpose){
    hipblasStatus_t status;
    hipblasHandle_t handle;
    status = hipblasCreate(&handle);
    
    if (status != HIPBLAS_STATUS_SUCCESS){    
        printf("cublas error %s\n", _cudaGetErrorEnum(status));
        return;
    }

    float alpha = 1.0, beta = 0.0;
    auto start = std::chrono::high_resolution_clock::now();
    if (no_transpose)
        status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                             N, M, K, 
                            &alpha, B, N, 
                            A, K, 
                            &beta, C, N);
    else{
        status = hipblasSgemm(handle, 
                             HIPBLAS_OP_T, HIPBLAS_OP_T,
                             M,
                             N, 
                             K,
                             &alpha,
                             A,
                             K,
                             B,
                             N,
                             &beta,
                             C,
                             M
                            );
    }
    hipDeviceSynchronize();
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start); 
    printf("hipblasSgemm (transpose: %s) kernel total elasped time: %ld ms \n", BOOL2STR( (!no_transpose) ), duration.count());
     
    if (status != HIPBLAS_STATUS_SUCCESS){    
        printf("cublas error %s\n", _cudaGetErrorEnum(status));
        return;
    }


    checkCudaErr( hipGetLastError() );
    hipblasDestroy(handle);
    return;
}
