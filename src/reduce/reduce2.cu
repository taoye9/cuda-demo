#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>


__global__ void col_sum(const float * __restrict__ din, float * __restrict__ dout, const int row, const int col){
    unsigned int tid = threadIdx.x;
    int offset = col >> 2;    

    const float4 *din_4 = reinterpret_cast<const float4 *>(din);
    float4 *dout_4 = reinterpret_cast<float4 *>(dout);
 
    float4 vdin;
    float4 sum4 = {0, 0, 0, 0};

    for (int i = 0; i < row; i ++){
        vdin = din_4[i * offset + tid]; 
        sum4.x += vdin.x;
        sum4.y += vdin.y;
        sum4.z += vdin.z;
        sum4.w += vdin.w;
    }  
    
    dout_4[tid] = sum4;
    return;
}


void init_data(const int row, const int col, float *hin){
    for (int y = 0; y < row; y ++) 
        for (int x = 0; x < col; x ++)
            hin[y * col + x] = (float) (x % 32);
}

void print_data(const int col, float *h){
        for (int x = 0; x < col; x ++)
            printf("%d : %f\n", x, h[x]);
}


void reduce(const int row, const int col){
    float *din, *dout, *hin, *hout; 
    hin = (float*)malloc(row * col * sizeof(float));
    hout = (float*)malloc(col * sizeof(float));
    
    init_data(row, col, hin);

    hipMalloc((void**)&din, row * col * sizeof(float));
    hipMalloc((void**)&dout, col * sizeof(float));
     
    hipMemcpy(din, hin, row * col * sizeof(float), hipMemcpyHostToDevice);
    
    assert(col % 4 == 0);

    dim3 block(col >> 2);
    dim3 grid(1);
    col_sum<<<grid, block>>>(din, dout, row, col);    
   
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
 
    hipMemcpy(hout, dout, col * sizeof(float), hipMemcpyDeviceToHost);
    print_data(col, hout);
    
    free(hin);
    free(hout);
    hipFree(din);
    hipFree(dout);
}

int main(){
    std::cout << "run reduce2 (16384, 512)" << std::endl;   
    reduce(16384, 512);

    return 0;
}
