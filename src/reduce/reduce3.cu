#include <hip/hip_runtime.h>
#include <iostream>

#define MAX_THREADS 1024
#define WARP_SIZE 32

/*  Tile method:
    1. blockDim.x == blockDim.y
    

*/
__global__ void col_sum(const float * __restrict__ din, float * __restrict__ dout, const int row, const int col){
    
    __shared__ float tile[WARP_SIZE][WARP_SIZE];

    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y_stride = col * blockDim.y;

    float sum = 0;

    if (idx < col) {
        unsigned int offset = threadIdx.y * col + idx;
        for (int r = threadIdx.y; r < row; r += blockDim.y){
            sum += din[offset];
            offset += y_stride;
        }
    }

    tile[threadIdx.x][threadIdx.y] = sum;
    __syncthreads();    

    if (threadIdx.x == 0) {
        sum = 0;
        for (int i = 0; i < blockDim.x; i ++)
            sum += tile[threadIdx.y][i];
        int pos = blockIdx.x * blockDim.x + threadIdx.y;
        if (pos < col){
            dout[pos] = sum;
        }
    }
    return;
}


void init_data(const int row, const int col, float *hin){
    for (int y = 0; y < row; y ++) 
        for (int x = 0; x < col; x ++)
            hin[y * col + x] = (float) (x % 32);
}

void print_data(const int col, float *h){
        for (int x = 0; x < col; x ++)
            printf("%d : %f\n", x, h[x]);
}


void reduce(const int row, const int col){
    float *din, *dout, *hin, *hout; 
    hin = (float*)malloc(row * col * sizeof(float));
    hout = (float*)malloc(col * sizeof(float));
    
    init_data(row, col, hin);

    hipMalloc((void**)&din, row * col * sizeof(float));
    hipMalloc((void**)&dout, col * sizeof(float));
     
    hipMemcpy(din, hin, row * col * sizeof(float), hipMemcpyHostToDevice);
    
    dim3 block(WARP_SIZE, WARP_SIZE);
    dim3 grid((col - 1) / WARP_SIZE + 1); //ceil(col / WARP_SIZE)

    col_sum<<<grid, block>>>(din, dout, row, col);    
   
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
 
    hipMemcpy(hout, dout, col * sizeof(float), hipMemcpyDeviceToHost);
    print_data(col, hout);
    
    free(hin);
    free(hout);
    hipFree(din);
    hipFree(dout);
}

int main(){
    std::cout << "run reduce1 (512, 1024)" << std::endl;   
    reduce(512, 1024);

    return 0;
}
